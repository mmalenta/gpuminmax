#include "hip/hip_runtime.h"
#include <algorithm>
#include <chrono>
#include <iostream>
#include <fstream>
#include <vector>

#include "errors.hpp"
#include "kernels.cuh"

int main(int argc, char *argv[]) {

    std::ifstream data("dataset.dat");

    float tmpvar = 0.0f;

    std::vector<float> data_vector;

    if (!data) {
        std::cout << "Could not open the data file\n";
        return 1;
    }

    std::cout << "Reading the data file...\n" << std::flush;
    while(data >> tmpvar) {
        data_vector.push_back(tmpvar);
    }

    std::cout << "Read the data file with " << data_vector.size() << " elements (" << data_vector.size() / 1024.0f / 1024.0f << "MiB)\n";
    std::cout << "STD min and max...\n" << std::flush;

    float stdmin = 0.0f;
    float stdmax = 0.0f;
    std::chrono::time_point<std::chrono::steady_clock> stdstart;
    std::chrono::time_point<std::chrono::steady_clock> stdend;

    stdstart = std::chrono::steady_clock::now();
    stdmin = *(std::min_element(data_vector.begin(), data_vector.end()));
    stdmax = *(std::max_element(data_vector.begin(), data_vector.end()));
    stdend = std::chrono::steady_clock::now();

    float stdelapsed = std::chrono::duration<float>(stdend - stdstart).count();
    std::cout << "Took " << stdelapsed << "s to obtain min and max using std\n";
    std::cout << "STD min: " << stdmin << ", STD max: " << stdmax << "\n" << std::flush;

    float *devicedata;
    cudaCheckError(hipMalloc((void**)&devicedata, data_vector.size() * sizeof(float)));

    dim3 grid(1, 1, 1);
    dim3 block(1, 1, 1);

    float gpumin = 0.0f;
    float gpumax = 0.0f;
    std::chrono::time_point<std::chrono::steady_clock> naivestart;
    std::chrono::time_point<std::chrono::steady_clock> naiveend;

    naivestart = std::chrono::steady_clock::now();
    naive_kernel<<<grid, block>>>(devicedata, gpumin, gpumax);
    hipDeviceSynchronize();
    cudaCheckError(hipGetLastError());
    naiveend = std::chrono::steady_clock::now();

    float naiveelapsed = std::chrono::duration<float>(naiveend - naivestart).count();
    std::cout << "Took " << naiveelapsed << "s to obtain min and max using simple GPU kernel\n";
    std::cout << "GPU (simple) min: " << gpumin << ", GPU (simple) max: " << gpumax << "\n" << std::flush;

    hipFree(devicedata);

    return 0;

}